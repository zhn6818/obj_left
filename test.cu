#include "hip/hip_runtime.h"
//
// Created by zhn68 on 2021/8/31.
//

#include <hip/hip_runtime.h>
#include ""
#include "test.h"

void print_array(int * array, int size)
{
    for (int i = 0; i < size; i++) {
        printf("%d ", array[i]);
    }
}
__global__ void increment_atomic(int * g)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    i = i % array_size;
    atomicAdd(&g[i], 1);
}
void incres()
{
    printf("%d total threads in %d blocks writing into %d arrays\n", num_threads, num_threads / block_width, array_size);

    int h_array[array_size];
    const int array_bytes = array_size * sizeof(int);

    int * d_array;
    hipMalloc((void **)&d_array, array_bytes);
    hipMemset((void *)d_array, 0, array_bytes);

//    timer.Start();
    increment_atomic << <num_threads / block_width, block_width >> >(d_array);
//    timer.Stop();

    hipMemcpy(h_array, d_array, array_bytes, hipMemcpyDeviceToHost);
    print_array(h_array, array_size);
//    printf("\nTime elapsed = %g ms\n", timer.Elapsed());
    hipFree(d_array);
}